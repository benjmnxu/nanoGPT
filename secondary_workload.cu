#include <iostream>
#include <stdexcept>
#include <csignal>

// For system calls
#include <unistd.h>

// For shared memory (IPC)
#include <sys/ipc.h>
#include <sys/shm.h>

// For CUDA and cuBLAS
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Helper to check for CUDA errors
#define CUDA_CHECK(call)                                                  \
    do {                                                                  \
        hipError_t err = call;                                           \
        if (err != hipSuccess) {                                         \
            fprintf(stderr, "CUDA Error at %s:%d -> %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err));                             \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    } while (0)

// Global pointers for cleanup
volatile int* shared_flag = nullptr;
float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
hipblasHandle_t cublas_handle;

// Signal handler for clean shutdown
void signalHandler(int signum) {
    std::cout << "\nSecondary workload shutting down..." << std::endl;
    
    // Detach shared memory
    if (shared_flag) {
        shmdt(shared_flag);
    }

    // Free GPU memory
    if (d_A) hipFree(d_A);
    if (d_B) hipFree(d_B);
    if (d_C) hipFree(d_C);

    // Destroy cuBLAS handle
    if (cublas_handle) hipblasDestroy(cublas_handle);

    exit(signum);
}

int main(int argc, char* argv[]) {
    // --- Configuration ---
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " --gpu-id <id>" << std::endl;
        return 1;
    }
    int gpu_id = std::stoi(argv[2]);

    // Matrix dimensions for the GEMM. Larger values = more power.
    // Tune these values to get the desired power draw on your H100.
    // Using powers of 2 is common for performance.
    const int M = 8192;
    const int N = 8192;
    const int K = 8192;

    // --- Setup Signal Handler ---
    signal(SIGINT, signalHandler);
    signal(SIGTERM, signalHandler);

    // --- Setup Shared Memory ---
    // The key must be identical to the one used in controller.cpp
    key_t key = ftok("firefly_ipc_key", gpu_id);
    if (key == -1) {
        perror("ftok");
        return 1;
    }

    // Get the existing shared memory segment created by the controller.
    int shmid = shmget(key, sizeof(int), 0666);
    if (shmid == -1) {
        perror("shmget");
        return 1;
    }

    // Attach to the segment.
    shared_flag = (int*)shmat(shmid, (void*)0, 0);
    if (shared_flag == (int*)(-1)) {
        perror("shmat");
        return 1;
    }
    std::cout << "Secondary workload for GPU " << gpu_id << " attached to shared memory." << std::endl;

    // --- Setup CUDA and cuBLAS ---
    CUDA_CHECK(hipSetDevice(gpu_id));

    // Allocate matrices on the GPU
    CUDA_CHECK(hipMalloc((void**)&d_A, sizeof(float) * M * K));
    CUDA_CHECK(hipMalloc((void**)&d_B, sizeof(float) * K * N));
    CUDA_CHECK(hipMalloc((void**)&d_C, sizeof(float) * M * N));

    // Initialize cuBLAS
    hipblasCreate(&cublas_handle);
    
    // GEMM calculation constants
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // --- Main Workload Loop ---
    std::cout << "Waiting for signal from controller..." << std::endl;
    while(true) {
        // Check the flag set by the controller.
        if (*shared_flag == 1) {
            // If flag is 1, launch the GEMM kernel to burn power.
            hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        M, N, K,
                        &alpha,
                        d_A, M,
                        d_B, K,
                        &beta,
                        d_C, M);
        } else {
            // If flag is 0, do nothing. A small sleep prevents this loop
            // from needlessly consuming CPU cycles (busy-waiting).
            usleep(1000); // Sleep for 1ms
        }
    }

    signalHandler(0);
    return 0;
}

#include <iostream>
#include <stdexcept>
#include <csignal>

// For system calls
#include <unistd.h>

// For shared memory (IPC)
#include <sys/ipc.h>
#include <sys/shm.h>

// For CUDA and cuBLAS
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Helper to check for CUDA errors
#define CUDA_CHECK(call)                                                  \
    do {                                                                  \
        hipError_t err = call;                                           \
        if (err != hipSuccess) {                                         \
            fprintf(stderr, "CUDA Error at %s:%d -> %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(err));                             \
            exit(EXIT_FAILURE);                                           \
        }                                                                 \
    } while (0)

// Global pointers for cleanup
volatile int* shared_flag = nullptr;
float *d_A = nullptr, *d_B = nullptr, *d_C = nullptr;
hipblasHandle_t cublas_handle;

// Signal handler for clean shutdown
void signalHandler(int signum) {
    std::cout << "\nSecondary workload shutting down..." << std::endl;
    
    // Detach shared memory
    if (shared_flag) {
        // --- FIX IS HERE ---
        // We explicitly cast away the 'volatile' qualifier for the same reason
        // as in the controller.
        shmdt(const_cast<int*>(shared_flag));
    }

    // Free GPU memory
    if (d_A) hipFree(d_A);
    if (d_B) hipFree(d_B);
    if (d_C) hipFree(d_C);

    // Destroy cuBLAS handle
    if (cublas_handle) hipblasDestroy(cublas_handle);

    exit(signum);
}

int main(int argc, char* argv[]) {
    // --- Configuration ---
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " --gpu-id <id>" << std::endl;
        return 1;
    }
    int gpu_id = std::stoi(argv[2]);

    const int M = 8192;
    const int N = 8192;
    const int K = 8192;

    // --- Setup Signal Handler ---
    signal(SIGINT, signalHandler);
    signal(SIGTERM, signalHandler);

    // --- Setup Shared Memory ---
    key_t key = ftok("firefly_ipc_key", gpu_id);
    if (key == -1) {
        perror("ftok");
        return 1;
    }

    int shmid = shmget(key, sizeof(int), 0666);
    if (shmid == -1) {
        perror("shmget");
        return 1;
    }

    shared_flag = (int*)shmat(shmid, (void*)0, 0);
    if (shared_flag == (int*)(-1)) {
        perror("shmat");
        return 1;
    }
    std::cout << "Secondary workload for GPU " << gpu_id << " attached to shared memory." << std::endl;

    // --- Setup CUDA and cuBLAS ---
    CUDA_CHECK(hipSetDevice(gpu_id));

    CUDA_CHECK(hipMalloc((void**)&d_A, sizeof(float) * M * K));
    CUDA_CHECK(hipMalloc((void**)&d_B, sizeof(float) * K * N));
    CUDA_CHECK(hipMalloc((void**)&d_C, sizeof(float) * M * N));

    hipblasCreate(&cublas_handle);
    
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // --- Main Workload Loop ---
    std::cout << "Waiting for signal from controller..." << std::endl;
    while(true) {
        if (*shared_flag == 1) {
            hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        M, N, K,
                        &alpha,
                        d_A, M,
                        d_B, K,
                        &beta,
                        d_C, M);
        } else {
            usleep(1000);
        }
    }

    signalHandler(0);
    return 0;
}

